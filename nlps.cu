void fft_plan_create()
{
    if(hipfftPlan3d(&plan_C2R, Nz, Nx, Ny, HIPFFT_C2R) != HIPFFT_SUCCESS) {
        printf("plan_C2R creation failed. Don't trust results. \n");
    };
    if(hipfftPlan3d(&plan_R2C, Nz, Nx, Ny, HIPFFT_R2C) != HIPFFT_SUCCESS) {
        printf("plan_R2C creation failed. Don't trust results. \n");
    }
}

void fft_plan_destroy()
{
    if(hipfftDestroy(plan_C2R) != HIPFFT_SUCCESS) printf("plan_C2R destruction failed. \n");
    if(hipfftDestroy(plan_R2C) != HIPFFT_SUCCESS) printf("plan_R2C destruction failed. \n");
}

void NLPS(hipComplex *result, hipComplex *f, hipComplex *g)
{
    GRADIENT (f, dx, dy);

    if(hipfftExecC2R(plan_C2R, dy, fdyR) != HIPFFT_SUCCESS) printf("fdyR calculation failed. \n");	
    if(hipfftExecC2R(plan_C2R, dx, fdxR) != HIPFFT_SUCCESS) printf("fdxR calculation failed. \n");

    GRADIENT (g, dx, dy);

    if(hipfftExecC2R(plan_C2R, dy, gdyR) != HIPFFT_SUCCESS) printf("gdyR calculation failed.  \n");
    if(hipfftExecC2R(plan_C2R, dx, gdxR) != HIPFFT_SUCCESS) printf("gdyR calculation failed.  \n");

    // Reuse fdxR as result 
    bracket <<<dG,dB>>> (fdxR, fdxR, fdyR, gdxR, gdyR, 1.0);

    if(hipfftExecR2C(plan_R2C, fdxR, result) != HIPFFT_SUCCESS) printf("R2C failed. \n");  

    scale <<<dG,dB>>> (result,1.0f/((float) Nx*Ny*Nz));

    // Dealias
    mask <<<dG,dB>>> (result);
}


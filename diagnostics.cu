#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////
// Alfven diagnostics
//////////////////////////////////////////////////////////////////////

// kz- Kperp spectra of Alfven waves

void energy_kz_kperp(hipComplex* kPhi, hipComplex* kA, float time, int jstep, struct NetCDF_ids id)
{

    //  int ikpmax = (int) ceil( sqrt( pow((float)(Nx-1)/3, 2) + pow((float)(Ny-1)/3, 2) ) );
    //  float kpmax = ceil( sqrt( pow((float)(Nx-1)/3, 2) + pow((float)(Ny-1)/3, 2) ) );

    int ikpmax = (int) (Nx-1)/3;
    float kpmax = (float) (Nx-1)/3;
    float *kinEnergy_kp, *magEnergy_kp;

    float *totEnergy_h, *kinEnergy_h, *magEnergy_h;

    totEnergy_h = (float*) malloc(sizeof(float));
    kinEnergy_h = (float*) malloc(sizeof(float));
    magEnergy_h = (float*) malloc(sizeof(float));


    kinEnergy_h[0] = 0.;
    magEnergy_h[0] = 0.;

    hipComplex *kPhi_h;
    kPhi_h = (hipComplex*) malloc(Nkc);
    CP_TO_CPU(kPhi_h, kPhi, Nkc);
    int Nyc = (Ny/2+1);
    // want to pull kphi up here to the host and sum it
    for (int j=0; j<Ny/2+1; j++) {
        for (int i=0; i<Nx; i++) {
            for (int k=0; k<Nz; k++) {
                int idx = j+i*Nyc+k*Nx*Nyc;
                //	if(kPhi_h[idx].x>0.1) {printf("Element %d\t%d\t%d\t Value %g \n",i,j,k,kPhi_h[idx].x);}
                kinEnergy_h[0] = kinEnergy_h[0] + kPhi_h[idx].x;	
            }
        }
    }
    free(kPhi_h);
    printf("Kinetic Energy = %g \n", kinEnergy_h[0]);


    // Allocate arrays to hold kinetic and magnetic energy vs k on GPU
    hipMalloc((void**) &kinEnergy_kp, sizeof(float)*ikpmax*Nz);
    hipMalloc((void**) &magEnergy_kp, sizeof(float)*ikpmax*Nz);

    // Set array values to zero
    zero <<<Nz,ikpmax>>> (kinEnergy_kp, Nz*ikpmax,1,1);
    zero <<<Nz,ikpmax>>> (magEnergy_kp, Nz*ikpmax,1,1);

    //loop through the ky's
    for(int ikp=1; ikp<ikpmax; ikp++) {
        kz_kpshellsum <<<dG, dB>>> (kPhi, ikp, kinEnergy_kp);
        kz_kpshellsum <<<dG, dB>>> (kA, ikp, magEnergy_kp);
    }


    CUDA_DEBUG("kz_kpshellsum: %s\n");

    float *kinEnergy_kp_h, *magEnergy_kp_h;

    // Allocate arrays to hold kinetic and magnetic energy vs k on CPU
    kinEnergy_kp_h = (float*) malloc(sizeof(float)*ikpmax*Nz);
    magEnergy_kp_h = (float*) malloc(sizeof(float)*ikpmax*Nz);

    // Set array values to zero
    for (int i=0; i<ikpmax*Nz; i++) {
        kinEnergy_kp_h[i] = 0.;
        magEnergy_kp_h[i] = 0.;
    }

    CP_TO_CPU (kinEnergy_kp_h, kinEnergy_kp, sizeof(float)*ikpmax*Nz);
    CUDA_DEBUG("Copying shell sums KE: %s\n");
    CP_TO_CPU (magEnergy_kp_h, magEnergy_kp, sizeof(float)*ikpmax*Nz);
    CUDA_DEBUG("Copying shell sums ME: %s\n");


    size_t start[3], count[3];
    start[0] = jstep;
    start[1] = 0;
    start[2] = 0;

    count[0] = 1;
    count[1] = ikpmax-1;
    count[2] = Nz;

    int retval;

    if (retval = nc_put_vara(id.file, id.b2, start, count, magEnergy_kp_h)) ERR(retval);
    if (retval = nc_put_vara(id.file, id.v2, start, count, kinEnergy_kp_h)) ERR(retval);
    if (retval = nc_sync(id.file)) ERR(retval);

    kinEnergy_h[0] = 0.;
    magEnergy_h[0] = 0.;

    for (int i=0; i<ikpmax*Nz; i++) {
        kinEnergy_h[0] = kinEnergy_h[0] + kinEnergy_kp_h[i];
        magEnergy_h[0] = magEnergy_h[0] + magEnergy_kp_h[i];
    }
    totEnergy_h[0] = kinEnergy_h[0] + magEnergy_h[0];

    printf("Total Energy = %g\t Kin Energy = %g\t Magnetic Energy = %g\n", 
            totEnergy_h[0], kinEnergy_h[0], magEnergy_h[0]);

    free(totEnergy_h); free(kinEnergy_h); free(magEnergy_h); 
    hipFree(kinEnergy_kp); hipFree(magEnergy_kp);
    free(kinEnergy_kp_h); free(magEnergy_kp_h);

}

////////////////////////////////////////
// Total energy
void energy(hipComplex* kPhi, hipComplex* kA, float time, int jstep, struct NetCDF_ids id)
{
    DEBUGPRINT("Entering energy\n");

    hipComplex *padded;
    hipMalloc((void**) &padded, sizeof(hipComplex)*Nx*Ny*Nz);

    hipComplex *totEnergy_h, *kinEnergy_h, *magEnergy_h;

    totEnergy_h = (hipComplex*) malloc(sizeof(hipComplex));
    kinEnergy_h = (hipComplex*) malloc(sizeof(hipComplex));
    magEnergy_h = (hipComplex*) malloc(sizeof(hipComplex));

    kinEnergy_h[0].x=0.;
    kinEnergy_h[0].y=0.;

    // integrate kA to find magnetic energy
    //sumReduc(magEnergy_h, kA, padded);
    sumReduc_gen(magEnergy_h, kA, padded, Nx, Ny, Nz);
    CUDA_DEBUG("sumreduc kA: %s\n");

    // integrate kPhi to find kinetic energy
    //sumReduc(kinEnergy_h, kPhi, padded);
    sumReduc_gen(kinEnergy_h, kPhi, padded, Nx, Ny, Nz);
    CUDA_DEBUG("sumreduc kPhi: %s\n");

    //calculate total energy
    totEnergy_h[0].x = kinEnergy_h[0].x + magEnergy_h[0].x;

    hipFree(padded);

    printf("Total Energy = %g\t Kin Energy = %g\t Magnetic Energy = %g\n", 
            totEnergy_h[0].x, kinEnergy_h[0].x, magEnergy_h[0].x);

    size_t start[1],count[1];
    start[0] = jstep;
    count[0] = 1;
    if (retval = nc_put_vara(id.file, id.b2_tot, start, count, &magEnergy_h[0].x )) ERR(retval);
    if (retval = nc_put_vara(id.file, id.v2_tot, start, count, &kinEnergy_h[0].x )) ERR(retval);
    if (retval = nc_sync(id.file)) ERR(retval);

    free(totEnergy_h); free(kinEnergy_h); free(magEnergy_h); 

    DEBUGPRINT("Exiting energy\n");

}    

void peak(hipComplex* kPhi, float time)
{
    float *rPhi;
    hipMalloc((void**) &rPhi, sizeof(float)*Nx*Ny*Nz);

    if(hipfftExecC2R(plan_C2R, kPhi, rPhi) != HIPFFT_SUCCESS) printf("oops in peak \n");

    float *rPhi_h;
    rPhi_h = (float*) malloc(sizeof(float)*Nx*Ny*Nz);
    CP_TO_CPU(rPhi_h, rPhi, sizeof(float)*Nx*Ny*Nz);
    hipFree(rPhi);

    int idx;
    int i, j, k, isave, jsave, ksave;
    int idxmax;
    float pmax, zpos;
    pmax = 0.;
    isave = 0;
    jsave = 0;
    ksave = 0;

    // find the peak 
    for (k=0; k<Nz; k++) {
        for (i=0; i<1; i++) {
            for (j=0; j<1; j++) {
                idx = j + Ny*i + Ny*Nx*k;
                if (rPhi_h[idx]>pmax)
                {
                    pmax = rPhi_h[idx];
                    isave = i;
                    jsave = j;
                    ksave = k;
                }
            }
        }
    }

    if (nsteps == 0) {
        fprintf (awp_collfile, "# z        Phi \n");
        for (k=0; k<Nz; k++) {
            zpos = k*2.*3.1415*Z0/Nz;
            for (i=0; i<1; i++) {
                for (j=0; j<1; j++) {
                    idx = j + Ny*i + Ny*Nx*k;
                    fprintf (awp_collfile, "%f \t %f \n", zpos, rPhi_h[idx]);
                }
            }
        }
    } else {
        zpos = ksave*2.*3.1415*Z0/Nz;
        fprintf (awp_collfile, "t= %f\t z= %f\t Peak= %f \n", time, zpos, pmax);
    }

    free (rPhi_h);  
}

// Assumes we are passed A (not k_perp A or somesuch )
void j_z_diag( hipComplex * A, float time, int jstep, struct NetCDF_ids id )
{
	return;
}


//////////////////////////////////////////////////////////////////////
// Main diagnostic calling functions
//////////////////////////////////////////////////////////////////////



void alf_diagnostics(hipComplex* kPhi, hipComplex* kA, hipComplex* zp, hipComplex* zm, float time, int jstep, struct NetCDF_ids id){
    // Calculate kperp**2 * phi and kperp**2 A for all alfven diagnostics
    addsubt <<<dG,dB>>> (kPhi, zp, zm, 1);
    //kPhi = zp+zm

    scale <<<dG,dB>>> (kPhi, .5);
    //kPhi = .5*(zp+zm) = phi

    addsubt <<<dG,dB>>> (kA, zp, zm, -1);
    //kA = zp-zm

    scale <<<dG,dB>>> (kA, .5);
    //kA = .5*(zp-zm) = A

    if (linonly) peak(kPhi, time);

    squareComplex <<<dG,dB>>> (kPhi);
    //kPhi = phi**2

    squareComplex <<<dG,dB>>> (kA);
    //kA = A**2

    fixFFT <<<dG,dB>>>(kPhi);
    fixFFT <<<dG,dB>>>(kA);

    multKPerp <<<dG,dB>>> (kPhi, kPhi, -1);
    //kPhi = (kperp**2) * (phi**2)

    multKPerp <<<dG,dB>>> (kA, kA, -1);
    //kA = (kperp**2) * (A**2)

    int retval;
    size_t start[1], count[1];
    start[0] = jstep; 
    count[0] = 1;
    if (retval = nc_put_vara(id.file, id.t, start, count, &time)) ERR(retval);

    energy_kz_kperp(kPhi, kA, time, jstep, id);
    energy(kPhi, kA, time, jstep, id);

    /*

    // Alfven wave collision diagnostics
    CP_ON_GPU(kPhi, zp, Nkc);
    CP_ON_GPU(kA, zm, Nkc);

    squareComplex <<<dG,dB>>> (kPhi);
    //kPhi = z+^2

    squareComplex <<<dG,dB>>> (kA);
    //kA = z-^2

    // No fixfft needed. If you called fixfft here, then the ky=0 mode needs to be
    // dealt with specially in the aw_coll_diag subroutine 
    aw_coll_diag(kPhi, kA, time);
     */

}

//////////////////////////////////////////////////////////////////////
// Initialize NetCDF diagnostics
//////////////////////////////////////////////////////////////////////

struct NetCDF_ids init_netcdf_diag(struct NetCDF_ids id){

    char str[255];

    int retval;

    //  int ikpmax = (int) ceil( sqrt( pow((float)(Nx-1)/3, 2) + pow((float)(Ny-1)/3, 2) ) );
    //  float kpmax = ceil( sqrt( pow((float)(Nx-1)/3, 2) + pow((float)(Ny-1)/3, 2) ) );
    int ikpmax = (int) (Nx-1)/3;
    float kpmax = (float) (Nx-1)/3;
    float kpar[Nz], kperp[ikpmax];

    strcpy(str, runname);
    strcat(str, ".nc");

    if (retval = nc_create(str, NC_CLOBBER, &id.file)) ERR(retval);

    if (retval = nc_def_dim(id.file, "nkperp", ikpmax-1,      &id.kperp_dim)) ERR(retval);
    if (retval = nc_def_dim(id.file, "nkz",    Nz,            &id.kz_dim))    ERR(retval);
    if (retval = nc_def_dim(id.file, "nkpar",  Nz,            &id.kpar_dim))  ERR(retval);
    if (retval = nc_def_dim(id.file, "time",   NC_UNLIMITED,  &id.t_dim))     ERR(retval);

    if (retval = nc_def_dim(id.file, "kx",   Nx,      &id.kx_dim))     ERR(retval);
    if (retval = nc_def_dim(id.file, "ky",   Ny/2+1,  &id.ky_dim))     ERR(retval);


    static char title[] = "Gandalf simulation data";
    if (retval = nc_put_att_text(id.file, NC_GLOBAL, "Title", strlen(title), title)) ERR(retval);

    time_t time_now;
    char* time_txt;

    time_now = time(NULL);
    time_txt = ctime(&time_now);

    if (retval = nc_put_att_text(id.file, NC_GLOBAL, "Date", strlen(time_txt)-1, time_txt)) ERR(retval);

    if (retval = nc_def_var(id.file, "GPU", NC_INT, 0, 0, &id.gpu)) ERR(retval);
    if (retval = nc_def_var(id.file, "restart", NC_INT, 0, 0, &id.restart)) ERR(retval);

    if (retval = nc_def_var(id.file, "Nx", NC_INT, 0, 0, &id.nx)) ERR(retval);
    if (retval = nc_def_var(id.file, "Ny", NC_INT, 0, 0, &id.ny)) ERR(retval);
    if (retval = nc_def_var(id.file, "Nz", NC_INT, 0, 0, &id.nz)) ERR(retval);

    if (retval = nc_def_var(id.file, "x0", NC_FLOAT, 0, 0, &id.x0)) ERR(retval);
    if (retval = nc_def_var(id.file, "y0", NC_FLOAT, 0, 0, &id.y0)) ERR(retval);
    if (retval = nc_def_var(id.file, "z0", NC_FLOAT, 0, 0, &id.z0)) ERR(retval);

    if (retval = nc_def_var(id.file, "nsteps", NC_INT, 0, 0, &id.nsteps)) ERR(retval);
    if (retval = nc_def_var(id.file, "t", NC_FLOAT, 1, &id.t_dim, &id.t)) ERR(retval);
    // should put text attribute "Time"
    // should put text attribute "Units"

    if (retval = nc_def_var(id.file, "nwrite", NC_INT, 0, 0, &id.nwrite)) ERR(retval);
    if (retval = nc_def_var(id.file, "nforce", NC_INT, 0, 0, &id.nforce)) ERR(retval);
    if (retval = nc_def_var(id.file, "maxdt", NC_FLOAT, 0, 0, &id.maxdt)) ERR(retval);
    if (retval = nc_def_var(id.file, "cfl", NC_FLOAT, 0, 0, &id.cfl)) ERR(retval);
    //  if (retval = nc_def_var(id.file, "restart_name", NC_CHAR, ?, ?, &restart_name)) ERR(retval);

    if (retval = nc_def_var(id.file, "alpha_z", NC_INT, 0, 0, &id.alpha_z)) ERR(retval);
    if (retval = nc_def_var(id.file, "alpha_hyper", NC_INT, 0, 0, &id.alpha_hyper)) ERR(retval);

    if (retval = nc_def_var(id.file, "nu_kz", NC_FLOAT, 0, 0, &id.nu_kz)) ERR(retval);
    if (retval = nc_def_var(id.file, "nu_hyper", NC_FLOAT, 0, 0, &id.nu_hyper)) ERR(retval);

    if (retval = nc_def_var(id.file, "kperp", NC_FLOAT, 1, &id.kperp_dim, &id.kperp)) ERR(retval);
    //  if (retval = nc_def_var(id.file, "kz",    NC_FLOAT, 1, &id.kz_dim,    &id.kz))    ERR(retval);
    if (retval = nc_def_var(id.file, "kpar",  NC_FLOAT, 1, &id.kpar_dim,  &id.kpar))  ERR(retval);

    if (retval = nc_def_var(id.file, "kpeak", NC_INT, 0, 0, &id.kpeak)) ERR(retval);

    if (driven) {
        if (retval = nc_def_dim(id.file, "nkstir",  nkstir, &id.stir_dim)) ERR(retval);
        if (retval = nc_def_var(id.file, "fampl", NC_FLOAT, 0, 0, &id.fampl)) ERR(retval);

        id.kstir[0] = id.stir_dim;

        if (retval = nc_def_var(id.file, "kstir_x", NC_INT, 1, id.kstir, &id.kstir_x)) ERR(retval);
        if (retval = nc_def_var(id.file, "kstir_y", NC_INT, 1, id.kstir, &id.kstir_y)) ERR(retval);
        if (retval = nc_def_var(id.file, "kstir_z", NC_INT, 1, id.kstir, &id.kstir_z)) ERR(retval);
    }

    id.kperps[0] = id.t_dim;
    id.kperps[1] = id.kperp_dim;

    id.kpars[0] = id.t_dim;
    id.kpars[1] = id.kpar_dim;

    id.kparperp[0] = id.t_dim;
    id.kparperp[1] = id.kperp_dim;
    id.kparperp[2] = id.kpar_dim;

    if (retval = nc_def_var(id.file, "b2_kparkperp", NC_FLOAT, 3, id.kparperp, &id.b2)) ERR(retval);
    if (retval = nc_def_var(id.file, "v2_kparkperp", NC_FLOAT, 3, id.kparperp, &id.v2)) ERR(retval);

    if (retval = nc_def_var(id.file, "v2", NC_FLOAT, 1, &id.t_dim, &id.v2_tot )) ERR(retval);
    if (retval = nc_def_var(id.file, "b2", NC_FLOAT, 1, &id.t_dim, &id.b2_tot )) ERR(retval);

    if (retval = nc_enddef(id.file)) ERR(retval);

    // need to define a temporary array for kpar (which will also serve for kz because kz is defined only as a function)

    for (int ikz=0; ikz<Nz; ikz++) {kpar[ikz] = kz(ikz);}
    for (int ikp=0; ikp<ikpmax; ikp++) {kperp[ikp] = ((float) ikp/ikpmax)*kpmax;} // only makes sense if X0 and Y0 each are unity. BD
    if (retval = nc_put_var(id.file, id.kpar,  kpar))  ERR(retval); 
    if (retval = nc_put_var(id.file, id.kperp, kperp)) ERR(retval); 

	 float kx_vals[Nx],ky_vals[ Ny/2 + 1 ];
	 for (int ikx=0; ikx < Nx; ++ikx) { kx_vals[ ikx ] = kx( ikx ); };
	 for (int ikx=0; ikx < Ny/2 + 1; ++ikx) { ky_vals[ ikx ] = ky( ikx ); };
    if (retval = nc_put_var(id.file, id.kx_dim, kx_vals))  ERR(retval); 
    if (retval = nc_put_var(id.file, id.ky_dim, ky_vals)) ERR(retval); 

    if (retval = nc_put_var(id.file, id.nx, &Nx)) ERR(retval);
    if (retval = nc_put_var(id.file, id.ny, &Ny)) ERR(retval);
    if (retval = nc_put_var(id.file, id.nz, &Nz)) ERR(retval);

    if (retval = nc_put_var(id.file, id.x0, &X0)) ERR(retval);
    if (retval = nc_put_var(id.file, id.y0, &Y0)) ERR(retval);
    if (retval = nc_put_var(id.file, id.z0, &Z0)) ERR(retval);

    if (retval = nc_put_var(id.file, id.nsteps, &nsteps)) ERR(retval);
    if (retval = nc_put_var(id.file, id.nwrite, &nwrite)) ERR(retval);
    if (retval = nc_put_var(id.file, id.nforce, &nforce)) ERR(retval);
    if (retval = nc_put_var(id.file, id.maxdt, &maxdt)) ERR(retval);
    if (retval = nc_put_var(id.file, id.cfl, &cfl)) ERR(retval);

    if (retval = nc_put_var(id.file, id.alpha_z, &alpha_z)) ERR(retval);
    if (retval = nc_put_var(id.file, id.alpha_hyper, &alpha_hyper)) ERR(retval);

    if (retval = nc_put_var(id.file, id.nu_kz, &nu_kz)) ERR(retval);
    if (retval = nc_put_var(id.file, id.nu_hyper, &nu_hyper)) ERR(retval);

    if (retval = nc_put_var(id.file, id.kpeak, &kpeak)) ERR(retval);

    if (driven) {
        if (retval = nc_put_var(id.file, id.nkstir, &nkstir)) ERR(retval);
        if (retval = nc_put_var(id.file, id.fampl, &fampl)) ERR(retval);

        size_t s[1];
        size_t c[1];

        s[0] = 0;
        c[0] = nkstir;

        if (retval = nc_put_vara(id.file, id.kstir_x, s, c, &kstir_x[0])) ERR(retval);
        if (retval = nc_put_vara(id.file, id.kstir_y, s, c, &kstir_y[0])) ERR(retval);
        if (retval = nc_put_vara(id.file, id.kstir_z, s, c, &kstir_z[0])) ERR(retval);
    }

    return id;
}


//////////////////////////////////////////////////////////////////////
// Initialize diagnostics
//////////////////////////////////////////////////////////////////////
void init_diag(){

    char str[255];

    /////////////////////////////////////////////////////////////////
    //Alfven diagnostics
    /////////////////////////////////////////////////////////////////
    //  strcpy(str, runname);
    //  strcat(str, ".energy");
    //  energyfile = fopen( str, "w+");
    //  strcpy(str, runname);
    //  strcat(str, ".speckzkp");
    //  alf_kzkpfile = fopen( str, "w+");

    /*  
    /////////////////////////////////////////////////////////////////
    //Alfven wave collision diagnostic
    /////////////////////////////////////////////////////////////////
    strcpy(str, runname);
    strcat(str, ".awcollp");
    awp_collfile = fopen( str, "w+");

    strcpy(str, runname);
    strcat(str, ".awcollm");
    awm_collfile = fopen( str, "w+");
     */

    strcpy(str, runname);
    strcat(str, ".awcollp");
    awp_collfile = fopen( str, "w+");
}
//////////////////////////////////////////////////////////////////////
void close_diag(){

    //  fclose(energyfile);
    //  fclose(alf_kzkpfile);

    fclose(awp_collfile);
    //  fclose(awm_collfile);


}

void close_netcdf_diag(struct NetCDF_ids id){

    int retval;

    if (retval= nc_close(id.file)) ERR(retval);

}

void diagnostics(hipComplex* zp, hipComplex* zm, float time, int jstep, struct NetCDF_ids id){

    alf_diagnostics(temp1, temp2, zp, zm, time, jstep, id);
}
//////////////////////////////////////////////////////////////////////
// Alfven collision diagnostic
//////////////////////////////////////////////////////////////////////
void aw_coll_diag(hipComplex* zp2, hipComplex* zm2, float time){

    int ikx, iky, ikz, index;

    hipComplex *energy;
    energy = (hipComplex*) malloc(sizeof(hipComplex)*2); 

    // Primary mode (1, 0, -1) copied over to energy[0]
    ikx = 1; iky=0; ikz = Nz-1;
    index = iky+(Ny/2+1)*ikx + (Ny/2+1)*Nx*ikz;
    CP_TO_CPU(energy, zp2 + index, sizeof(hipComplex));

    // Primary mode (0, 1, 1) copied over to energy[1]
    ikx = 0; iky=1; ikz = 1;
    index = iky+(Ny/2+1)*ikx + (Ny/2+1)*Nx*ikz;
    CP_TO_CPU(energy+1, zp2 + index, sizeof(hipComplex));

    // Primary modes written out. The ky=0 mode gets scaled up due to reality condition
    fprintf(awp_collfile, "%g \t %g \t", time, sqrt(energy[0].x+energy[1].x));

    // Secondary mode (1, 1, 0) copied over to energy[0]
    ikx = 1; iky=1; ikz = 0;
    index = iky+(Ny/2+1)*ikx + (Ny/2+1)*Nx*ikz;
    CP_TO_CPU(energy, zp2 + index, sizeof(hipComplex));

    //Secondary mode written out
    fprintf(awp_collfile, "%g \t ", sqrt(energy[0].x));

    // Tertiary mode (2, 1, -1) copied over to energy[0]
    ikx = 2; iky=1; ikz = Nz-1;
    index = iky+(Ny/2+1)*ikx + (Ny/2+1)*Nx*ikz;
    CP_TO_CPU(energy, zp2 + index, sizeof(hipComplex));

    // Tertiary mode (1, 2, 1) copied over to energy[1]
    ikx = 1; iky=2; ikz = 1;
    index = iky+(Ny/2+1)*ikx + (Ny/2+1)*Nx*ikz;
    CP_TO_CPU(energy+1, zp2 + index, sizeof(hipComplex));

    // Tertiary mode written out
    fprintf(awp_collfile, "%g\n", sqrt(energy[0].x + energy[1].x));

    fflush(awp_collfile);

    // Primary mode (1, 0, -1) copied over to energy[0]
    ikx = 1; iky=0; ikz = Nz-1;
    index = iky+(Ny/2+1)*ikx + (Ny/2+1)*Nx*ikz;
    CP_TO_CPU(energy, zm2 + index, sizeof(hipComplex));

    // Primary mode (0, 1, 1) copied over to energy[1]
    ikx = 0; iky=1; ikz = 1;
    index = iky+(Ny/2+1)*ikx + (Ny/2+1)*Nx*ikz;
    CP_TO_CPU(energy+1, zm2 + index, sizeof(hipComplex));

    // Primary modes written out. The ky=0 mode gets scaled up due to reality condition
    fprintf(awm_collfile, "%g \t %g \t", time, sqrt(energy[0].x+energy[1].x));

    // Secondary mode (1, 1, 0) copied over to energy[0]
    ikx = 1; iky=1; ikz = 0;
    index = iky+(Ny/2+1)*ikx + (Ny/2+1)*Nx*ikz;
    CP_TO_CPU(energy, zm2 + index, sizeof(hipComplex));

    //Secondary mode written out
    fprintf(awm_collfile, "%g \t ", sqrt(energy[0].x));

    // Tertiary mode (2, 1, -1) copied over to energy[0]
    ikx = 2; iky=1; ikz = Nz-1;
    index = iky+(Ny/2+1)*ikx + (Ny/2+1)*Nx*ikz;
    CP_TO_CPU(energy, zm2 + index, sizeof(hipComplex));

    // Tertiary mode (1, 2, 1) copied over to energy[1]
    ikx = 1; iky=2; ikz = 1;
    index = iky+(Ny/2+1)*ikx + (Ny/2+1)*Nx*ikz;
    CP_TO_CPU(energy+1, zm2 + index, sizeof(hipComplex));

    // Tertiary mode written out
    fprintf(awm_collfile, "%g\n", sqrt(energy[0].x + energy[1].x));

    fflush(awm_collfile);

    free(energy);


}
